#include "Hatrix/util/handle.h"

#include "Hatrix/classes/Matrix.h"


namespace Hatrix {

  hipblasHandle_t blasH = nullptr;
  hipsolverHandle_t solvH = nullptr;

  void init() {
    hipblasCreate(&blasH);
    hipsolverDnCreate(&solvH);
  }

  void terminate() {
    hipblasDestroy(blasH); blasH = 0;
    hipsolverDnDestroy(solvH); blasH = 0;
  }


} // namespace Hatrix
