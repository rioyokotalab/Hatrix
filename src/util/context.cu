#include "Hatrix/util/context.h"

#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include "hip/hip_runtime_api.h"
#include "hipblas.h"
#include "hipsolver.h"
#include "hiprand.h"

namespace Hatrix {
  
  size_t Context::nStreams = 0;
  size_t Context::workspaceInBytesOnDevice = 0;
  size_t Context::workspaceInBytesOnHost = 0;
  
  hipStream_t* Context::stream = nullptr;
  hipblasHandle_t* Context::cublasH = nullptr;
  hipsolverHandle_t* Context::cusolverH = nullptr;
  hipsolverDnParams_t* Context::cusolverParams = nullptr;
  hiprandGenerator_t* Context::curandH = nullptr;
  
  void** Context::bufferOnDevice = nullptr;
  void** Context::bufferOnHost = nullptr;
  int* Context::info = nullptr;
  size_t Context::sid = 0;
  bool Context::forking = false;
  
  void Context::init(int argc, const char** argv) {
    if (nStreams > 0)
      Context::finalize();
  
    Context::nStreams = (size_t)(argc > 1 ? strtoull(argv[1], nullptr, 0) : 1);
    Context::workspaceInBytesOnDevice = (size_t)(argc > 2 ? strtoull(argv[2], nullptr, 0) : DEFAULT_LWORK);
    Context::workspaceInBytesOnHost = (size_t)(argc > 3 ? strtoull(argv[3], nullptr, 0) : DEFAULT_LWORK_HOST);
  
    Context::stream = new hipStream_t [Context::nStreams];
    Context::cublasH = new hipblasHandle_t [Context::nStreams];
    Context::cusolverH = new hipsolverHandle_t [Context::nStreams];
    Context::cusolverParams = new hipsolverDnParams_t [Context::nStreams];
    Context::curandH = new hiprandGenerator_t [Context::nStreams];
  
    Context::bufferOnHost = new void* [Context::nStreams];
    Context::bufferOnDevice = new void* [Context::nStreams];
  
    for (size_t i = 0; i < Context::nStreams; i++) {
      hipStreamCreateWithFlags(Context::stream + i, hipStreamDefault);
      hipblasCreate(Context::cublasH + i);
      hipblasSetStream(Context::cublasH[i], Context::stream[i]);
  
      if (Context::workspaceInBytesOnDevice)
        hipMalloc(reinterpret_cast<void**>(Context::bufferOnDevice + i), Context::workspaceInBytesOnDevice);
      if (Context::workspaceInBytesOnHost)
        Context::bufferOnHost[i] = (void*)malloc(Context::workspaceInBytesOnHost);
  
      hipsolverDnCreate(Context::cusolverH + i);
      hipsolverSetStream(Context::cusolverH[i], Context::stream[i]);
      hipsolverDnCreateParams(Context::cusolverParams + i);
  
      hiprandCreateGenerator(Context::curandH + i, HIPRAND_RNG_PSEUDO_DEFAULT);
      hiprandSetStream(Context::curandH[i], Context::stream[i]);
    }
  
    hipMalloc(reinterpret_cast<void**>(&Context::info), Context::nStreams * sizeof(int));
    Context::sid = 0;
    Context::forking = false;
  }
  
  void Context::finalize() {
    for (size_t i = 0; i < Context::nStreams; i++) {
      if (Context::stream)
        hipStreamDestroy(Context::stream[i]);
      if (Context::cublasH)
        hipblasDestroy(Context::cublasH[i]);
      if (Context::cusolverH)
        hipsolverDnDestroy(Context::cusolverH[i]);
      if (Context::cusolverParams[i])
        hipsolverDnDestroyParams(Context::cusolverParams[i]);
      if (Context::curandH)
        hiprandDestroyGenerator(Context::curandH[i]);
      if (Context::bufferOnDevice[i])
        hipFree(Context::bufferOnDevice[i]);
      if (Context::bufferOnHost[i])
        free(Context::bufferOnHost[i]);
    }
  
    Context::nStreams = 0;
    Context::workspaceInBytesOnDevice = 0;
    Context::workspaceInBytesOnHost = 0;
  
    if (Context::stream)
      delete[] Context::stream;
    if (Context::cublasH)
      delete[] Context::cublasH;
    if (Context::cusolverH)
      delete[] Context::cusolverH;
    if (Context::cusolverParams)
      delete[] Context::cusolverParams;
    if (Context::curandH)
      delete[] Context::curandH;
  
    Context::stream = nullptr;
    Context::cublasH = nullptr;
    Context::cusolverH = nullptr;
    Context::cusolverParams = nullptr;
    Context::curandH = nullptr;
  
    if (Context::bufferOnDevice)
      delete[] Context::bufferOnDevice;
    if (Context::bufferOnHost)
      delete[] Context::bufferOnHost;
    if (Context::info)
      hipFree(Context::info);
  
    bufferOnDevice = nullptr;
    bufferOnHost = nullptr;
    info = nullptr;
  }
  
  void Context::join() {
    Context::sid = 0;
    Context::forking = false;
    hipDeviceSynchronize();
  }
  
  void Context::fork() {
    Context::forking = true;
  }

  void Context::critical() {
    Context::forking = false;
  }
  
  void Context::iterate() {
    if (Context::forking)
      Context::sid = Context::sid == Context::nStreams - 1 ? 0 : Context::sid + 1;
  }
  

}  // namespace Hatrix
