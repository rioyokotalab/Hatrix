#include "Hatrix/util/context.h"

#include "hipblas.h"
#include "hipsolver.h"

#include "Hatrix/classes/Matrix.h"

namespace Hatrix {

hipblasHandle_t blasH = nullptr;
hipsolverHandle_t solvH = nullptr;

void init() {
  hipblasCreate(&blasH);
  hipsolverDnCreate(&solvH);
}

void terminate() {
  hipblasDestroy(blasH);
  blasH = 0;
  hipsolverDnDestroy(solvH);
  blasH = 0;
}

void sync() { hipDeviceSynchronize(); }

}  // namespace Hatrix
