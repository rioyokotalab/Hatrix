#include "Hatrix/functions/blas.h"

#include "hipblas.h"
#include <cassert>

#include "Hatrix/classes/Matrix.h"

namespace Hatrix {

void matmul(const Matrix &A, const Matrix &B, Matrix &C, bool transA,
            bool transB, double alpha, double beta) {
  void* args[1];
  runtime_args(args, arg_t::BLAS);
  hipblasHandle_t blasH = reinterpret_cast<hipblasHandle_t>(args[0]);
  hipblasDgemm(blasH, transA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
              transB ? HIPBLAS_OP_T : HIPBLAS_OP_N, C.rows, C.cols,
              transA ? A.rows : A.cols, &alpha, &A, A.rows, &B, B.rows, &beta,
              &C, C.rows);

};

void triangular_matmul(const Matrix& A, Matrix& B, int side, int uplo,
                       bool transA, bool diag, double alpha) {
  assert(side == Left ? (transA ? A.rows == B.rows : A.cols == B.rows)
                      : (transA ? B.cols == A.cols : B.cols == A.rows));
  void* args[1];
  runtime_args(args, arg_t::BLAS);
  hipblasHandle_t blasH = reinterpret_cast<hipblasHandle_t>(args[0]);
  hipblasDtrmm(blasH, side == Left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT,
              uplo == Upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER,
	      transA ? HIPBLAS_OP_T : HIPBLAS_OP_N, 
	      diag ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT, B.rows, B.cols, &alpha, 
	      &A, A.rows, &B, B.rows, &B, B.rows);
}

void solve_triangular(const Matrix &A, Matrix &B, int side, int uplo, bool diag,
                      bool transA, double alpha) {
  void* args[1];
  runtime_args(args, arg_t::BLAS);
  hipblasHandle_t blasH = reinterpret_cast<hipblasHandle_t>(args[0]);
  hipblasDtrsm(blasH, side == Left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT,
              uplo == Upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER,
              transA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
              diag ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT, B.rows, B.cols,
              &alpha, &A, A.rows, &B, B.rows);
}

void scale(Matrix& A, double alpha) {
  void* args[1];
  runtime_args(args, arg_t::BLAS);
  hipblasHandle_t blasH = reinterpret_cast<hipblasHandle_t>(args[0]);
  hipblasDscal(blasH, A.rows * A.cols, &alpha, &A, 1);
}

} // namespace Hatrix
