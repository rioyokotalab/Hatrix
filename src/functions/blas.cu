#include "Hatrix/functions/blas.h"

#include "hipblas.h"
#include <cassert>

#include "Hatrix/classes/Matrix.h"
#include "Hatrix/util/context.h"

namespace Hatrix {

void matmul(const Matrix &A, const Matrix &B, Matrix &C, bool transA,
            bool transB, double alpha, double beta) {
  hipblasHandle_t handle = Context::cublasH[Context::sid];
  Context::iterate();
  hipblasDgemm(handle, transA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
              transB ? HIPBLAS_OP_T : HIPBLAS_OP_N, C.rows, C.cols,
              transA ? A.rows : A.cols, &alpha, &A, A.rows, &B, B.rows, &beta,
              &C, C.rows);

};

Matrix matmul(const Matrix& A, const Matrix& B, bool transA, bool transB,
              double alpha) {
  Matrix C(transA ? A.cols : A.rows, transB ? B.rows : B.cols);
  matmul(A, B, C, transA, transB, alpha, 0);
  return C;
}

void triangular_matmul(const Matrix& A, Matrix& B, Side side, Mode uplo,
                       bool transA, bool diag, double alpha) {
  assert(side == Left ? (transA ? A.rows == B.rows : A.cols == B.rows)
                      : (transA ? B.cols == A.cols : B.cols == A.rows));
  hipblasHandle_t handle = Context::cublasH[Context::sid];
  Context::iterate();
  hipblasDtrmm(handle, side == Left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT,
              uplo == Upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER,
	      transA ? HIPBLAS_OP_T : HIPBLAS_OP_N, 
	      diag ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT, B.rows, B.cols, &alpha, 
	      &A, A.rows, &B, B.rows, &B, B.rows);
}

void solve_triangular(const Matrix &A, Matrix &B, Side side, Mode uplo, bool diag,
                      bool transA, double alpha) {
  hipblasHandle_t handle = Context::cublasH[Context::sid];
  Context::iterate();
  hipblasDtrsm(handle, side == Left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT,
              uplo == Upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER,
              transA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
              diag ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT, B.rows, B.cols,
              &alpha, &A, A.rows, &B, B.rows);
}

void scale(Matrix& A, double alpha) {
  void* args[1];
  hipblasHandle_t handle = Context::cublasH[Context::sid];
  Context::iterate();
  hipblasDscal(handle, A.rows * A.cols, &alpha, &A, 1);
}

} // namespace Hatrix
