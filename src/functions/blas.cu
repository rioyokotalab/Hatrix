#include "Hatrix/functions/blas.h"

#include "hipblas.h"
#include "hipsolver.h"
#include <cassert>

#include "Hatrix/classes/Matrix.h"

namespace Hatrix {

extern hipblasHandle_t blasH;
extern hipsolverHandle_t solvH;

void matmul(const Matrix &A, const Matrix &B, Matrix &C, bool transA,
            bool transB, double alpha, double beta) {
  hipblasDgemm(blasH, transA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
              transB ? HIPBLAS_OP_T : HIPBLAS_OP_N, C.rows, C.cols,
              transA ? A.rows : A.cols, &alpha, &A, A.rows, &B, B.rows, &beta,
              &C, C.rows);

  hipDeviceSynchronize();
};

void triangular_matmul(const Matrix& A, Matrix& B, int side, int uplo,
                       bool transA, bool diag, double alpha) {
  assert(side == Left ? (transA ? A.rows == B.rows : A.cols == B.rows)
                      : (transA ? B.cols == A.cols : B.cols == A.rows));
  hipblasDtrmm(blasH, side == Left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT,
              uplo == Upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER,
	      transA ? HIPBLAS_OP_T : HIPBLAS_OP_N, 
	      diag ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT, B.rows, B.cols, &alpha, 
	      &A, A.rows, &B, B.rows, &B, B.rows);
}

void solve_triangular(const Matrix &A, Matrix &B, int side, int uplo, bool diag,
                      bool transA, double alpha) {
  hipblasDtrsm(blasH, side == Left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT,
              uplo == Upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER,
              transA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
              diag ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT, B.rows, B.cols,
              &alpha, &A, A.rows, &B, B.rows);

  hipDeviceSynchronize();
}

void scale(Matrix& A, double alpha) {
  hipblasDscal(blasH, A.rows * A.cols, &alpha, &A, 1);
  hipDeviceSynchronize();
}

} // namespace Hatrix
