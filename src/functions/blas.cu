#include "Hatrix/functions/blas.h"

#include "Hatrix/classes/Matrix.h"

#include "Hatrix/handle.h"
#include "hipblas.h"
#include "hipsolver.h"

namespace Hatrix {

  hipblasHandle_t blasH = nullptr;
  hipsolverHandle_t solvH = nullptr;

  void init() {
    hipblasCreate(&blasH);
    hipsolverDnCreate(&solvH);
  }

  void terminate() {
    hipblasDestroy(blasH); blasH = 0;
    hipsolverDnDestroy(solvH); blasH = 0;
  }

void matmul(
  const Matrix& A, const Matrix& B, Matrix& C,
  bool transA, bool transB, double alpha, double beta
) {
  hipblasDgemm(
    blasH,
    transA ? HIPBLAS_OP_T : HIPBLAS_OP_N, transB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
    C.rows, C.cols, transA ? A.rows : A.cols,
    &alpha, &A, A.rows, &B, B.rows,
    &beta, &C, C.rows
  );

  hipDeviceSynchronize();
};

void solve_triangular(
  const Matrix& A, Matrix& B,
  int side, int uplo, bool diag, bool transA, double alpha
) {
  hipblasDtrsm(
    blasH,
    side == Left ? HIPBLAS_SIDE_LEFT :  HIPBLAS_SIDE_RIGHT,
    uplo == Upper ? HIPBLAS_FILL_MODE_UPPER :  HIPBLAS_FILL_MODE_LOWER,
    transA ? HIPBLAS_OP_T : HIPBLAS_OP_N, diag ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT,
    B.rows, B.cols,
    &alpha, &A, A.rows, &B, B.rows
  );

  hipDeviceSynchronize();
}

} // namespace Hatrix
