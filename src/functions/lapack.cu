#include "hip/hip_runtime.h"
#include "Hatrix/functions/lapack.h"

#include <algorithm>
#include <cassert>

#include "hipsolver.h"

#include "Hatrix/classes/Matrix.h"

namespace Hatrix {

extern hipblasHandle_t blasH;
extern hipsolverHandle_t solvH;

void lu(Matrix &A, Matrix &L, Matrix &U) {
  int Lwork;
  hipsolverDnDgetrf_bufferSize(solvH, A.rows, A.cols, &A, A.rows, &Lwork);

  double *work;
  hipMalloc(reinterpret_cast<void **>(&work), Lwork);

  hipsolverDnDgetrf(solvH, A.rows, A.cols, &A, A.rows, work, nullptr, nullptr);

  hipDeviceSynchronize();
  hipFree(work);

  for (int i = 0; i < L.cols && i < A.cols; i++) {
    double one = 1;
    hipMemcpy(&L + i * L.rows + i, &one, sizeof(double),
               hipMemcpyHostToDevice);
    if (i + 1 < A.rows)
      hipMemcpy(&L + i * L.rows + i + 1, &A + i * A.rows + i + 1,
                 (A.rows - i - 1) * sizeof(double), hipMemcpyDeviceToDevice);
  }

  for (int i = 0; i < A.cols; i++) {
    hipMemcpy(&U + i * U.rows, &A + i * A.rows,
               std::min(i + 1, (int)A.rows) * sizeof(double),
               hipMemcpyDeviceToDevice);
  }
}

void qr(Matrix &A, Matrix &Q, Matrix &R) {
  int Lwork;
  double *tau, *work;

  hipsolverDnDgeqrf_bufferSize(solvH, A.rows, A.cols, &A, A.rows, &Lwork);

  hipMalloc(reinterpret_cast<void **>(&work), Lwork);
  hipMalloc(reinterpret_cast<void **>(&tau),
             std::min(A.rows, A.cols) * sizeof(double));
  hipsolverDnDgeqrf(solvH, A.rows, A.cols, &A, A.rows, tau, work, Lwork,
                   nullptr);

  hipDeviceSynchronize();
  hipMemcpy(&Q, &A, Q.rows * std::min(A.cols, Q.cols) * sizeof(double),
             hipMemcpyDeviceToDevice);

  for (int i = 0; i < A.cols; i++) {
    hipMemcpy(&R + i * R.rows, &A + i * A.rows,
               std::min(i + 1, (int)A.rows) * sizeof(double),
               hipMemcpyDeviceToDevice);
  }
  hipsolverDnDorgqr(solvH, Q.rows, Q.cols, Q.cols, &Q, Q.rows, tau, work, Lwork,
                   nullptr);

  hipDeviceSynchronize();
  hipFree(tau);
  hipFree(work);
}

void svd(Matrix &A, Matrix &U, Matrix &S, Matrix &V) {
  double *work, *s;
  hipMallocManaged(reinterpret_cast<void **>(&s),
                    std::min(S.rows, S.cols) * sizeof(double));

  int Lwork;

  hipsolverDnDgesvd_bufferSize(solvH, A.rows, A.cols, &Lwork);
  hipMalloc(reinterpret_cast<void **>(&work), Lwork);

  hipsolverDnDgesvd(solvH, 'S', 'S', A.rows, A.cols, &A, A.rows, s, &U, U.rows,
                   &V, V.rows, work, Lwork, nullptr, nullptr);

  hipDeviceSynchronize();

  for (int i = 0; i < std::min(S.rows, S.cols); i++) {
    S(i, i) = s[i];
  }

  hipFree(s);
  hipFree(work);
}

double truncated_svd(Matrix &A, Matrix &U, Matrix &S, Matrix &V, int64_t rank) {
  char jobu = 'S', jobv = 'S';
  int64_t m = A.rows, n = A.cols, iters = 2;
  int64_t lda = A.rows, ldu = U.rows, ldv = V.rows;

  rank = rank > m ? m : rank;
  rank = rank > n ? n : rank;
  p = p < 0 ? 0 : p;
  p = p + rank > m ? m - rank : p;
  p = p + rank > n ? n - rank : p;

  hipsolverDnParams_t params_gesvdr;
  hipsolverDnCreateParams(&params_gesvdr);

  size_t workspaceInBytesOnDevice_gesvdr, workspaceInBytesOnHost_gesvdr;
  cusolverDnXgesvdr_bufferSize(solvH, params_gesvdr, jobu, jobv, m, n, rank, p, iters, HIP_R_64F, (void*)A, lda, HIP_R_64F, S,
    HIP_R_64F, U, ldu, HIP_R_64F, V, ldv, HIP_R_64F, &workspaceInBytesOnDevice_gesvdr, &workspaceInBytesOnHost_gesvdr);
  
  double* Work_host = (double*)malloc(workspaceInBytesOnHost_gesvdr), *Work_dev;
  hipMalloc(reinterpret_cast<void **>(&Work_dev), Lwork);

  cusolverDnXgesvdr(solvH, params_gesvdr, jobu, jobv, m, n, rank, p, iters, HIP_R_64F, (void*)A, lda, HIP_R_64F, S, 
    HIP_R_64F, U, ldu, HIP_R_64F, V, ldv, HIP_R_64F, Work_dev, workspaceInBytesOnDevice_gesvdr, Work_host, workspaceInBytesOnHost_gesvdr, nullptr);

  free(Work_host);
  hipFree(Work_dev);
  hipsolverDnDestroyParams(params_gesvdr);
  return 0.;
}

double norm(const Matrix& A) {
  double result;
  hipblasDnrm2(blasH, A.rows * A.cols, &A, 1, &result);
  hipDeviceSynchronize();
  return result;
}

}  // namespace Hatrix
