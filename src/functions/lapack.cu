#include "hip/hip_runtime.h"
#include "Hatrix/classes/Matrix.h"

#include "Hatrix/util/handle.h"
#include "hipsolver.h"
#include <algorithm>

namespace Hatrix {

  extern hipblasHandle_t blasH;
  extern hipsolverHandle_t solvH;

void lu(Matrix& A, Matrix& L, Matrix& U) {

  int Lwork;
  hipsolverDnDgetrf_bufferSize(solvH, A.rows, A.cols, &A, A.rows, &Lwork);

  double* work;
  hipMalloc(reinterpret_cast<void**>(&work), Lwork);

  hipsolverDnDgetrf(solvH, A.rows, A.cols, &A, A.rows, work, nullptr, nullptr);

  hipDeviceSynchronize();
  hipFree(work);

  for (int i = 0; i < L.cols && i < A.cols; i++) {
    double one = 1;
    hipMemcpy(&L + i * L.rows + i, &one, sizeof(double), hipMemcpyHostToDevice);
    if (i + 1 < A.rows)
      hipMemcpy(&L + i * L.rows + i + 1, &A + i * A.rows + i + 1, (A.rows - i - 1) * sizeof(double), hipMemcpyDeviceToDevice);
  }

  for (int i = 0; i < A.cols; i++) {
    hipMemcpy(&U + i * U.rows, &A + i * A.rows, std::min(i + 1, (int)A.rows) * sizeof(double), hipMemcpyDeviceToDevice);
  }

}

void qr(Matrix& A, Matrix& Q, Matrix& R) {

  int Lwork;
  double* tau, * work;

  hipsolverDnDgeqrf_bufferSize(solvH, A.rows, A.cols, &A, A.rows, &Lwork);

  hipMalloc(reinterpret_cast<void**>(&work), Lwork);
  hipMalloc(reinterpret_cast<void**>(&tau), std::min(A.rows, A.cols) * sizeof(double));
  hipsolverDnDgeqrf(solvH, A.rows, A.cols, &A, A.rows, tau, work, Lwork, nullptr);

  hipDeviceSynchronize();
  hipMemcpy(&Q, &A, Q.rows * std::min(A.cols, Q.cols) * sizeof(double), hipMemcpyDeviceToDevice);

  for (int i = 0; i < A.cols; i++) {
    hipMemcpy(&R + i * R.rows, &A + i * A.rows, std::min(i + 1, (int)A.rows) * sizeof(double), hipMemcpyDeviceToDevice);
  }
  hipsolverDnDorgqr(solvH, Q.rows, Q.cols, Q.cols, &Q, Q.rows, tau, work, Lwork, nullptr);

  hipDeviceSynchronize();
  hipFree(tau);
  hipFree(work);

}

void svd(Matrix& A, Matrix& U, Matrix& S, Matrix& V) {
  
  int Lwork;
  hipsolverDnDgesvd_bufferSize(solvH, A.rows, A.cols, &Lwork);
  double* work;

  hipMalloc(reinterpret_cast<void**>(&work), Lwork);
  hipsolverDnDgesvd(solvH, 'A', 'A', A.rows, A.cols, &A, A.rows, &S, &U, U.rows, &V, V.rows, work, Lwork, nullptr, nullptr);

  hipDeviceSynchronize();
  for (int i = std::min(S.rows, S.cols); i > 0; i--) {
    double zero = 0;
    hipMemcpy(&S + i * S.rows + i, &S + i, sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(&S + i, &zero, sizeof(double), hipMemcpyHostToDevice);
  }

  hipFree(work);
}

double truncated_svd(Matrix& A, Matrix& U, Matrix& S, Matrix& V, int64_t rank) {

  return 0;
}

} // namespace Hatrix
