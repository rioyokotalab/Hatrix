#include "hip/hip_runtime.h"
#include "Hatrix/functions/lapack.h"

#include <algorithm>
#include <cassert>

#include "hipsolver.h"

#include "Hatrix/classes/Matrix.h"

namespace Hatrix {

extern hipblasHandle_t blasH;
extern hipsolverHandle_t solvH;

void lu(Matrix &A, Matrix &L, Matrix &U) {
  int Lwork;
  hipsolverDnDgetrf_bufferSize(solvH, A.rows, A.cols, &A, A.rows, &Lwork);

  double *work;
  hipMalloc(reinterpret_cast<void **>(&work), Lwork);

  hipsolverDnDgetrf(solvH, A.rows, A.cols, &A, A.rows, work, nullptr, nullptr);

  hipDeviceSynchronize();
  hipFree(work);

  for (int i = 0; i < L.cols && i < A.cols; i++) {
    double one = 1;
    hipMemcpy(&L + i * L.rows + i, &one, sizeof(double),
               hipMemcpyHostToDevice);
    if (i + 1 < A.rows)
      hipMemcpy(&L + i * L.rows + i + 1, &A + i * A.rows + i + 1,
                 (A.rows - i - 1) * sizeof(double), hipMemcpyDeviceToDevice);
  }

  for (int i = 0; i < A.cols; i++) {
    hipMemcpy(&U + i * U.rows, &A + i * A.rows,
               std::min(i + 1, (int)A.rows) * sizeof(double),
               hipMemcpyDeviceToDevice);
  }
}

void qr(Matrix &A, Matrix &Q, Matrix &R) {
  int Lwork;
  double *tau, *work;

  hipsolverDnDgeqrf_bufferSize(solvH, A.rows, A.cols, &A, A.rows, &Lwork);

  hipMalloc(reinterpret_cast<void **>(&work), Lwork);
  hipMalloc(reinterpret_cast<void **>(&tau),
             std::min(A.rows, A.cols) * sizeof(double));
  hipsolverDnDgeqrf(solvH, A.rows, A.cols, &A, A.rows, tau, work, Lwork,
                   nullptr);

  hipDeviceSynchronize();
  hipMemcpy(&Q, &A, Q.rows * std::min(A.cols, Q.cols) * sizeof(double),
             hipMemcpyDeviceToDevice);

  for (int i = 0; i < A.cols; i++) {
    hipMemcpy(&R + i * R.rows, &A + i * A.rows,
               std::min(i + 1, (int)A.rows) * sizeof(double),
               hipMemcpyDeviceToDevice);
  }
  hipsolverDnDorgqr(solvH, Q.rows, Q.cols, Q.cols, &Q, Q.rows, tau, work, Lwork,
                   nullptr);

  hipDeviceSynchronize();
  hipFree(tau);
  hipFree(work);
}

void svd(Matrix &A, Matrix &U, Matrix &S, Matrix &V) {
  double *work, *s;
  hipMallocManaged(reinterpret_cast<void **>(&s),
                    std::min(S.rows, S.cols) * sizeof(double));

  int Lwork;

  hipsolverDnDgesvd_bufferSize(solvH, A.rows, A.cols, &Lwork);
  hipMalloc(reinterpret_cast<void **>(&work), Lwork);

  hipsolverDnDgesvd(solvH, 'S', 'S', A.rows, A.cols, &A, A.rows, s, &U, U.rows,
                   &V, V.rows, work, Lwork, nullptr, nullptr);

  hipDeviceSynchronize();

  for (int i = 0; i < std::min(S.rows, S.cols); i++) {
    S(i, i) = s[i];
  }

  hipFree(s);
  hipFree(work);
}

double truncated_svd(Matrix &A, Matrix &U, Matrix &S, Matrix &V, int64_t rank) {
  assert(rank < A.min_dim());
  svd(A, U, S, V);
  double expected_err = 0;
  for (int64_t k = rank; k < A.min_dim(); ++k)
    expected_err += S(k, k) * S(k, k);
  U.shrink(U.rows, rank);
  S.shrink(rank, rank);
  V.shrink(rank, V.cols);
  return expected_err;
}

double norm(const Matrix& A) {
  double result;
  hipblasDnrm2(blasH, A.rows * A.cols, &A, 1, &result);
  hipDeviceSynchronize();
  return result;
}

}  // namespace Hatrix
