#include "hip/hip_runtime.h"
#include "Hatrix/classes/Matrix.h"

#include "Hatrix/handle.h"
#include "hipsolver.h"

namespace Hatrix {

  extern hipblasHandle_t blasH;
  extern hipsolverHandle_t solvH;

void lu(Matrix& A, Matrix& L, Matrix& U) {

  int Lwork;
  hipsolverDnDgetrf_bufferSize(solvH, A.rows, A.cols, &A, A.rows, &Lwork);

  double* work;
  hipMalloc(reinterpret_cast<void**>(&work), Lwork);

  hipsolverDnDgetrf(solvH, A.rows, A.cols, &A, A.rows, work, nullptr, nullptr);

  // copy out U and L

  // copy out the rest of U if trapezoidal

  // L: set diagonal to 1 and upper triangular matrix to 0

  // U: set lower triangular to 0?

  hipDeviceSynchronize();
  hipFree(work);

}

void qr(const Matrix& A, Matrix& Q, Matrix& R) {

  int Lwork, k = std::min(A.rows, A.cols);
  double* tau, * work;

  hipblasDcopy(blasH, A.rows * A.cols, &A, 1, &Q, 1);
  hipsolverDnDgeqrf_bufferSize(solvH, Q.rows, Q.cols, &Q, Q.rows, &Lwork);

  hipMalloc(reinterpret_cast<void**>(&work), Lwork);
  hipMalloc(reinterpret_cast<void**>(&tau), k * sizeof(double));
  hipsolverDnDgeqrf(solvH, Q.rows, Q.cols, &Q, Q.rows, tau, work, Lwork, nullptr);

  double one = 1, zero = 0;
  hipblasDgeam(blasH, HIPBLAS_OP_N, HIPBLAS_OP_N, Q.rows, Q.cols, &one, &Q, Q.rows, &zero, &R, R.rows, &R, R.rows);
  hipsolverDnDorgqr(solvH, Q.rows, Q.cols, k, &Q, Q.rows, tau, work, Lwork, nullptr);

  hipDeviceSynchronize();
  hipFree(tau);
  hipFree(work);

}

} // namespace Hatrix
