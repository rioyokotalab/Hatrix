#include "hip/hip_runtime.h"
#include "Hatrix/functions/lapack.h"

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <tuple>

#include "hipblas.h"
#include "hipsolver.h"
#include "hip/hip_runtime_api.h"

#include "Hatrix/classes/Matrix.h"
#include "Hatrix/util/context.h"

namespace Hatrix {

void dgetrf(double* a, int64_t m, int64_t n, int64_t lda, int64_t* ipiv) {
  void* args[7];
  runtime_args(args, arg_t::SOLV);
  hipsolverHandle_t handle = reinterpret_cast<hipsolverHandle_t>(args[0]);
  hipsolverDnParams_t params = reinterpret_cast<hipsolverDnParams_t>(args[1]);
  void* work = args[2], * work_host = args[4];
  size_t Lwork = *reinterpret_cast<size_t*>(args[3]), Lwork_host = *reinterpret_cast<size_t*>(args[5]);
  int* dev_info = reinterpret_cast<int*>(args[6]);

  size_t workspaceInBytesOnDevice_getrf, workspaceInBytesOnHost_getrf;
  hipsolverDnXgetrf_bufferSize(handle, params, m, n, HIP_R_64F, a, lda, HIP_R_64F, &workspaceInBytesOnDevice_getrf, &workspaceInBytesOnHost_getrf);
  if (workspaceInBytesOnDevice_getrf <= Lwork && workspaceInBytesOnHost_getrf <= Lwork_host)
    hipsolverDnXgetrf(handle, params, m, n, HIP_R_64F, a, lda, ipiv, HIP_R_64F, work, Lwork, work_host, Lwork_host, dev_info);
  else
    fprintf(stderr, "Insufficient work for DGETRF. %zu, %zu\n", workspaceInBytesOnDevice_getrf, workspaceInBytesOnHost_getrf);
}

void dtricpy(int kind, int uplo, int diag, int64_t m, int64_t n, double* dst, int64_t ldd, const double* src, int64_t lds) {
  void* args[3];
  runtime_args(args, arg_t::STREAM);
  hipStream_t stream = reinterpret_cast<hipStream_t>(args[0]);

  lds = lds == 0 ? ldd : lds;
  bool diag_unit = static_cast<hipblasDiagType_t>(diag) == HIPBLAS_DIAG_UNIT;

  if (static_cast<hipblasFillMode_t>(uplo) == HIPBLAS_FILL_MODE_LOWER) {
    int64_t n_col = m - diag_unit;
    for (int64_t i = 0; i < n && n_col > 0; i++) {
      int64_t offset = m - n_col;
      hipMemcpyAsync(dst + i * ldd + offset, src + i * lds + offset, n_col * sizeof(double), static_cast<hipMemcpyKind>(kind), stream);
      n_col--;
    }
  }
  else if (static_cast<hipblasFillMode_t>(uplo) == HIPBLAS_FILL_MODE_UPPER) {
    int64_t n_col = 1;
    for (int64_t i = diag_unit; i < n; i++) {
      hipMemcpyAsync(dst + i * ldd, src + i * lds, n_col * sizeof(double), static_cast<hipMemcpyKind>(kind), stream);
      n_col = n_col == m ? m : n_col + 1;
    }
  }

  if (diag_unit) {
    double one = 1;
    for (int i = 0; i < m && i < n; i++)
      hipMemcpyAsync(dst + i * ldd + i, &one, sizeof(double), hipMemcpyHostToDevice, stream);
  }
}

void lu(Matrix &A, Matrix &L, Matrix &U) {
  mode_t old = parallel_mode(mode_t::SERIAL);
  dgetrf(&A, A.rows, A.cols, A.rows, nullptr);
  dtricpy(hipMemcpyDefault, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_DIAG_UNIT, A.rows, A.cols, &L, L.rows, &A, A.rows);
  parallel_mode(old);
  dtricpy(hipMemcpyDefault, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, A.rows, A.cols, &U, U.rows, &A, A.rows);
}

void qr(Matrix &A, Matrix &Q, Matrix &R) {

}

void dgesvd(int64_t m, int64_t n, double* A, int64_t lda, double* S, double* U, int64_t ldu, double* V, int64_t ldv) {
  void* args[7];
  runtime_args(args, arg_t::SOLV);
  hipsolverHandle_t handle = reinterpret_cast<hipsolverHandle_t>(args[0]);
  hipsolverDnParams_t params = reinterpret_cast<hipsolverDnParams_t>(args[1]);
  void* work = args[2], * work_host = args[4];
  size_t Lwork = *reinterpret_cast<size_t*>(args[3]), Lwork_host = *reinterpret_cast<size_t*>(args[5]);
  int* dev_info = reinterpret_cast<int*>(args[6]);

  auto jobz = CUSOLVER_EIGMODE_VECTOR;
  int econ = 1;
  double h_err_sigma;

  size_t workspaceInBytesOnDevice_gesvd, workspaceInBytesOnHost_gesvd;
  cusolverDnXgesvdp_bufferSize(handle, params, jobz, econ, m, n, HIP_R_64F, (void*)A, lda, HIP_R_64F, S,
    HIP_R_64F, U, ldu, HIP_R_64F, V, ldv, HIP_R_64F, &workspaceInBytesOnDevice_gesvd, &workspaceInBytesOnHost_gesvd);

  if (workspaceInBytesOnDevice_gesvd <= Lwork && workspaceInBytesOnHost_gesvd <= Lwork_host)
    cusolverDnXgesvdp(handle, params, jobz, econ, m, n, HIP_R_64F, (void*)A, lda, HIP_R_64F, S, 
      HIP_R_64F, U, ldu, HIP_R_64F, V, ldv, HIP_R_64F, work, Lwork, work_host, Lwork_host, dev_info, &h_err_sigma);
  else
    fprintf(stderr, "Insufficient work for DGESVDR. %zu, %zu\n", workspaceInBytesOnDevice_gesvdr, workspaceInBytesOnHost_gesvdr);
}

void dsv2m(double* s, int64_t m, int64_t n, int64_t lds) {
  void* args[3];
  runtime_args(args, arg_t::STREAM);
  hipStream_t stream = reinterpret_cast<hipStream_t>(args[0]);
  double* work = reinterpret_cast<double*>(args[1]);
  size_t Lwork = *reinterpret_cast<size_t*>(args[2]);

  int64_t r = m > n ? n : m;
  if (Lwork >= r * sizeof(double)) {
    hipMemcpyAsync(work, s, r * sizeof(double), hipMemcpyDeviceToDevice, stream);
    hipMemsetAsync(s, 0, sizeof(double) * lds * n, stream);
    for (int64_t i = 0; i < r; i++)
      hipMemcpyAsync(s + i * lds + i, work + i, sizeof(double), hipMemcpyDeviceToDevice, stream);
  }
  else
    fprintf(stderr, "Insufficient work for extending singular vector to matrix. %zu\n", r * sizeof(double));
}

void dvt2v(double* vt, int64_t m, int64_t n, int64_t ldvt, int64_t ldv) {
  void* args[3];
  runtime_args(args, arg_t::STREAM);
  hipStream_t stream = reinterpret_cast<hipStream_t>(args[0]);
  double* work = reinterpret_cast<double*>(args[1]);
  size_t Lwork = *reinterpret_cast<size_t*>(args[2]);

  ldvt = ldvt < m ? m : ldvt;
  ldv = ldv < n ? n : ldv;

  if (Lwork >= sizeof(double) * m * n) {
    for (int64_t i = 0; i < n; i++)
      for (int64_t j = 0; j < m; j++)
        hipMemcpyAsync(work + j * n + i, vt + i * ldvt + j, sizeof(double), hipMemcpyDeviceToDevice, stream);

    hipMemcpy2DAsync(vt, sizeof(double) * ldv, work, sizeof(double) * n, sizeof(double) * n, m, hipMemcpyDeviceToDevice, stream);
  }
  else
    fprintf(stderr, "Insufficient work for tranposing vt to v. %zu\n", sizeof(double) * m * n);
}


void svd(Matrix &A, Matrix &U, Matrix &S, Matrix &V) {
  mode_t old = parallel_mode(mode_t::SERIAL);
  int64_t r = A.rows > A.cols ? A.cols : A.rows;
  dgesvd(A.rows, A.cols, &A, A.rows, &S, &U, U.rows, &V, V.cols);
  dsv2m(&S, S.rows, S.cols, S.rows);
  parallel_mode(old);
  dvt2v(&V, V.cols, V.rows, V.cols, V.rows);
}

double truncated_svd(Matrix &A, Matrix &U, Matrix &S, Matrix &V, int64_t rank) {
  assert(rank < A.min_dim());
  svd(A, U, S, V);
  sync();
  double expected_err = 0;
  for (int64_t k = rank; k < A.min_dim(); ++k)
    expected_err += S(k, k) * S(k, k);
  U.shrink(U.rows, rank);
  S.shrink(rank, rank);
  V.shrink(rank, V.cols);
  return std::sqrt(expected_err);
}

std::tuple<Matrix, Matrix, Matrix, double> truncated_svd(Matrix& A,
                                                         int64_t rank) {
  Matrix U(A.rows, A.min_dim());
  Matrix S(A.min_dim(), A.min_dim());
  Matrix V(A.min_dim(), A.cols);
  double expected_err = truncated_svd(A, U, S, V, rank);
  return {std::move(U), std::move(S), std::move(V), expected_err};
}

double norm(const Matrix& A) {
  void* args[1];
  runtime_args(args, arg_t::BLAS);
  hipblasHandle_t blasH = reinterpret_cast<hipblasHandle_t>(args[0]);

  double result;
  hipblasDnrm2(blasH, A.rows * A.cols, &A, 1, &result);
  hipDeviceSynchronize();
  return result;
}

}  // namespace Hatrix
