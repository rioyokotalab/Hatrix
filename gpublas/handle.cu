
#include <handle.h>

#include <cstdlib>
#include <hip/hip_runtime_api.h>

using namespace Hatrix::gpu;

Stream::Stream(size_t Lwork, size_t Lwork_host) {
  Stream::Lwork = Lwork;
  Stream::Lwork_host = Lwork_host;

  hipStreamCreateWithFlags(&stream, hipStreamDefault);
  hipblasCreate(&cublasH);
  hipblasSetStream(cublasH, stream);
  hipMalloc(reinterpret_cast<void**>(&Workspace), Lwork);
  hipMalloc(reinterpret_cast<void**>(&info), sizeof(int));
  Work_host = (Lwork_host > 0) ? (void*)malloc(Lwork_host) : nullptr;

  //hipblasSetWorkspace(cublasH, Workspace, Lwork);
  hipsolverDnCreate(&cusolverH);
  hipsolverSetStream(cusolverH, stream);
  hipsolverDnCreateParams(&cusolverParams);
}

Stream::~Stream() {
  if (info)
    hipFree(info);
  if (Workspace)
    hipFree(Workspace);
  if (Work_host)
    free(Work_host);
  if (cusolverParams)
    hipsolverDnDestroyParams(cusolverParams);
  if (cusolverH)
    hipsolverDnDestroy(cusolverH);
  if (cublasH)
    hipblasDestroy(cublasH);
  if (stream)
    hipStreamDestroy(stream);
}

void Stream::sync() const {
  hipStreamSynchronize(stream);
}

Stream::operator hipStream_t() {
  return stream;
}

Stream::operator hipblasHandle_t() {
  return cublasH;
}

Stream::operator hipsolverHandle_t() {
  return cusolverH;
}

Stream::operator hipsolverDnParams_t() {
  return cusolverParams;
}

Stream::operator double*() {
  return (double*)Workspace;
}

Stream::operator float* () {
  return (float*)Workspace;
}

Stream::operator void* () {
  return (void*)Workspace;
}

Stream::operator size_t() {
  return Lwork;
}

Stream::operator int() {
  int i;
  hipMemcpy(&i, info, sizeof(int), hipMemcpyDeviceToHost);
  return i;
}

Stream::operator int* () {
  return info;
}
